#include "hip/hip_runtime.h"
#include "IConvolutionLayer.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <algorithm>

template <typename Dtype>
__global__ void im2col_gpu_kernel(const int n, const Dtype* data_im,
	const int height, const int width, const int kernel_h, const int kernel_w,
	const int pad_h, const int pad_w, const int stride_h, const int stride_w,
	const int dilation_h, const int dilation_w, const int height_col, const int width_col,
	Dtype* data_col)
{
	CUDA_KERNEL_LOOP(index, n)
	{
		const int h_index = index / width_col;
		const int h_col = h_index % height_col;
		const int w_col = index % width_col;
		const int c_im = h_index / height_col;
		const int c_col = c_im * kernel_h * kernel_w;
		const int h_offset = h_col * stride_h - pad_h;
		const int w_offset = w_col * stride_w - pad_w;
		Dtype* data_col_ptr = data_col;
		data_col_ptr += (c_col * height_col + h_col) * width_col + w_col;
		const Dtype* data_im_ptr = data_im;
		data_im_ptr += (c_im * height + h_offset) * width + w_offset;
		for (int i = 0; i < kernel_h; ++i)
		{
			for (int j = 0; j < kernel_w; ++j)
			{
				int h_im = h_offset + i * dilation_h;
				int w_im = w_offset + j * dilation_w;
				*data_col_ptr = (h_im >= 0 && w_im >= 0 && h_im < height && w_im < width) ?
					data_im_ptr[i * dilation_h * width + j * dilation_w] : 0;
				data_col_ptr += height_col * width_col;
			}
		}
	}
}

template <typename Dtype>
void CNN_Im2Col_GPU(const Dtype* data_im, const int channels,
	const int height, const int width, const int kernel_h, const int kernel_w,
	const int pad_h, const int pad_w, const int stride_h, const int stride_w,
	const int dilation_h, const int dilation_w, Dtype* data_col)
{
	// We are going to launch channels * height_col * width_col kernels, each
	// kernel responsible for copying a single-channel grid.
	int height_col = (height + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
	int width_col = (width + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;
	int num_kernels = channels * height_col * width_col;
	// NOLINT_NEXT_LINE(whitespace/operators)
	im2col_gpu_kernel<Dtype> << <CNN_GET_BLOCKS(num_kernels),
		CNN_CUDA_NUM_THREADS >> >(num_kernels, data_im, height, width, kernel_h, kernel_w, pad_h,
			pad_w, stride_h, stride_w, dilation_h, dilation_w, height_col, width_col, data_col);

#ifdef USE_GLOG
	CUDA_POST_KERNEL_CHECK;
#else
	if (hipSuccess != hipPeekAtLastError())
	{
		printf("im2col_gpu_kernel error:%d\n");
		return;
	}
#endif
}

template
void CNN_Im2Col_GPU<float>(const float* data_im, const int channels,
	const int height, const int width, const int kernel_h, const int kernel_w,
	const int pad_h, const int pad_w, const int stride_h, const int stride_w,
	const int dilation_h, const int dilation_w, float* data_col);
template
void CNN_Im2Col_GPU<double>(const double* data_im, const int channels,
	const int height, const int width, const int kernel_h, const int kernel_w,
	const int pad_h, const int pad_w, const int stride_h, const int stride_w,
	const int dilation_h, const int dilation_w, double* data_col);

namespace fecnn {
	// ��Ҫ�봦����������ͱ���һ��
#define sizeofStorageT 4
#define sizeofComputeT 4
#define CPUStorage2ComputeT(x) (x)
#define CPUCompute2StorageT(x) (x)
#define GPUStorage2ComputeT(x) (x)
#define GPUCompute2StorageT(x) (x)
#define GPUgemm hipblasSgemm
#define GPUasum hipblasSasum
#define ISNAN(x) (std::isnan(x)) //����ĳ���������Ƿ��� ������
#define ComputeT_MIN FLT_MIN //1.17549e-38

	void FatalError(const int lineNumber = 0) {
		std::cerr << "FatalError";
		// if (fileName!="") std::cerr<<" at FILE "<<fileName;
		if (lineNumber != 0) std::cerr << " at LINE " << lineNumber;
		std::cerr << ". Program Terminated." << std::endl;
		hipDeviceReset();
		exit(EXIT_FAILURE);
	}

	void checkCUDA(const char* fileName, const int lineNumber, hipError_t status) {
		if (status != hipSuccess) {
			std::cerr << "CUDA failure at FILE " << fileName << " at LINE " << lineNumber << ": " << status << std::endl;
			FatalError();
		}
	}

	void checkCUBLAS(const char* fileName, const int lineNumber, hipblasStatus_t status) {
		if (status != HIPBLAS_STATUS_SUCCESS) {
			std::cerr << "CUBLAS failure at FILE " << fileName << " at LINE " << lineNumber << ": ";
			switch (status) {
			case HIPBLAS_STATUS_SUCCESS:             std::cerr << "HIPBLAS_STATUS_SUCCESS" << std::endl; break;
			case HIPBLAS_STATUS_NOT_INITIALIZED:     std::cerr << "HIPBLAS_STATUS_NOT_INITIALIZED" << std::endl; break;
			case HIPBLAS_STATUS_ALLOC_FAILED:        std::cerr << "HIPBLAS_STATUS_ALLOC_FAILED" << std::endl; break;
			case HIPBLAS_STATUS_INVALID_VALUE:       std::cerr << "HIPBLAS_STATUS_INVALID_VALUE" << std::endl; break;
			case HIPBLAS_STATUS_ARCH_MISMATCH:       std::cerr << "HIPBLAS_STATUS_ARCH_MISMATCH" << std::endl; break;
			case HIPBLAS_STATUS_MAPPING_ERROR:       std::cerr << "HIPBLAS_STATUS_MAPPING_ERROR" << std::endl; break;
			case HIPBLAS_STATUS_EXECUTION_FAILED:    std::cerr << "HIPBLAS_STATUS_EXECUTION_FAILED" << std::endl; break;
			case HIPBLAS_STATUS_INTERNAL_ERROR:      std::cerr << "HIPBLAS_STATUS_INTERNAL_ERROR" << std::endl; break;
			case HIPBLAS_STATUS_NOT_SUPPORTED:       std::cerr << "HIPBLAS_STATUS_NOT_SUPPORTED" << std::endl; break;
			case HIPBLAS_STATUS_UNKNOWN:       std::cerr << "HIPBLAS_STATUS_UNKNOWN" << std::endl; break;
			}
			FatalError();
		}
		checkCUDA(fileName, lineNumber, hipGetLastError());
	}
	// CUDA: grid stride looping
#define CUDA_KERNEL_LOOP(i, n) \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
       i < (n); \
       i += blockDim.x * gridDim.x)

	// CUDA: use 512 threads per block
	const int FECNN_CUDA_NUM_THREADS = 512;

	// CUDA: number of blocks for threads.
	inline int FECNN_GET_BLOCKS(const int N) {
		return (N + FECNN_CUDA_NUM_THREADS - 1) / FECNN_CUDA_NUM_THREADS;
	}
	void fecnn_gpu_gemm(hipblasHandle_t cublasHandle, const hipblasOperation_t TransA,
		const hipblasOperation_t TransB, const int M, const int N, const int K,
		const StorageT alpha, const StorageT* A, const StorageT* B, const StorageT beta,
		StorageT* C) {
		// Note that cublas follows fortran order.
		int lda = (TransA == HIPBLAS_OP_N) ? K : M;
		int ldb = (TransB == HIPBLAS_OP_N) ? N : K;
		hipblasOperation_t cuTransA =
			(TransA == HIPBLAS_OP_N) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
		hipblasOperation_t cuTransB =
			(TransB == HIPBLAS_OP_N) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
		checkCUBLAS(__FILE__, __LINE__,
			hipblasSgemm(cublasHandle, cuTransB, cuTransA,
				N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
	}

	// @fixme ֻ�ʺϼ���float��
	void fecnn_gpu_gemv(hipblasHandle_t cublasHandle, const hipblasOperation_t TransA, const int M,
		const int N, const StorageT alpha, const StorageT* A, const StorageT* x,
		const StorageT beta, StorageT* y) {
		hipblasOperation_t cuTransA =
			(TransA == HIPBLAS_OP_N) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
		checkCUBLAS(__FILE__, __LINE__,
			hipblasSgemv(cublasHandle, cuTransA, N, M, &alpha,
				A, N, x, 1, &beta, y, 1));
	}

	__global__ void im2col_gpu_kernel(const int n, const StorageT* data_im,
		const int height, const int width, const int kernel_h, const int kernel_w,
		const int pad_h, const int pad_w,
		const int stride_h, const int stride_w,
		const int dilation_h, const int dilation_w,
		const int height_col, const int width_col,
		StorageT* data_col) {
		CUDA_KERNEL_LOOP(index, n) {
			const int h_index = index / width_col;
			const int h_col = h_index % height_col;
			const int w_col = index % width_col;
			const int c_im = h_index / height_col;
			const int c_col = c_im * kernel_h * kernel_w;
			const int h_offset = h_col * stride_h - pad_h;
			const int w_offset = w_col * stride_w - pad_w;

			StorageT* data_col_ptr = data_col;
			data_col_ptr += (c_col * height_col + h_col) * width_col + w_col;

			const StorageT* data_im_ptr = data_im;
			data_im_ptr += (c_im * height + h_offset) * width + w_offset;

			for (int i = 0; i < kernel_h; ++i) {
				for (int j = 0; j < kernel_w; ++j) {
					int h_im = h_offset + i * dilation_h;
					int w_im = w_offset + j * dilation_w;
					*data_col_ptr =
						(h_im >= 0 && w_im >= 0 && h_im < height && w_im < width) ?
						data_im_ptr[i * dilation_h * width + j * dilation_w] : 0;
					data_col_ptr += height_col * width_col;
				}
			}
		}
	}

	void im2col_gpu(const StorageT* data_im, const int channels,
		const int height, const int width, const int kernel_h, const int kernel_w,
		const int pad_h, const int pad_w,
		const int stride_h, const int stride_w,
		const int dilation_h, const int dilation_w,
		StorageT* data_col) {

		// std::cout<<"channels="<<channels<<" height="<<height<<" width="<<width<<std::endl; // conv1:[1,28,28] conv2:[20,12,12]
		// std::cout<<"dilation_h="<<dilation_h<<" dilation_w="<<dilation_w<<std::endl; // =1,1

		// We are going to launch channels * height_col * width_col kernels, each
		// kernel responsible for copying a single-channel grid.
		int height_col = (height + 2 * pad_h -
			(dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
		int width_col = (width + 2 * pad_w -
			(dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;
		int num_kernels = channels * height_col * width_col;

		// std::cout<<"height_col="<<height_col<<" width_col="<<width_col<<" channels="<<channels<<std::endl; // conv1:[24,24,1] ; conv2:[8,8,20]

		// NOLINT_NEXT_LINE(whitespace/operators)
		im2col_gpu_kernel << <FECNN_GET_BLOCKS(num_kernels),
			FECNN_CUDA_NUM_THREADS >> > (
				num_kernels, data_im, height, width, kernel_h, kernel_w, pad_h,
				pad_w, stride_h, stride_w, dilation_h, dilation_w, height_col,
				width_col, data_col);
	}

	__global__ void col2im_gpu_kernel(const int n, const StorageT* data_col,
		const int height, const int width, const int channels,
		const int kernel_h, const int kernel_w,
		const int pad_h, const int pad_w,
		const int stride_h, const int stride_w,
		const int dilation_h, const int dilation_w,
		const int height_col, const int width_col,
		StorageT* data_im)
	{
		CUDA_KERNEL_LOOP(index, n)
		{
			StorageT val = 0;
			const int w_im = index % width + pad_w;
			const int h_im = (index / width) % height + pad_h;
			const int c_im = index / (width * height);
			int kernel_extent_w = (kernel_w - 1) * dilation_w + 1;
			int kernel_extent_h = (kernel_h - 1) * dilation_h + 1;
			// compute the start and end of the output
			const int w_col_start = (w_im < kernel_extent_w) ? 0 : (w_im - kernel_extent_w) / stride_w + 1;
			int tmp = (int)(w_im / stride_w + 1);
			const int w_col_end = tmp < width_col ? tmp : width_col;
			const int h_col_start = (h_im < kernel_extent_h) ? 0 : (h_im - kernel_extent_h) / stride_h + 1;
			tmp = (int)(h_im / stride_h + 1);
			const int h_col_end = tmp < height_col ? tmp : height_col;
			// TODO: use LCM of stride and dilation to avoid unnecessary loops
			for (int h_col = h_col_start; h_col < h_col_end; h_col += 1) {
				for (int w_col = w_col_start; w_col < w_col_end; w_col += 1) {
					int h_k = (h_im - h_col * stride_h);
					int w_k = (w_im - w_col * stride_w);
					if (h_k % dilation_h == 0 && w_k % dilation_w == 0) {
						h_k /= dilation_h;
						w_k /= dilation_w;
						int data_col_index = (((c_im * kernel_h + h_k) * kernel_w + w_k) *
							height_col + h_col) * width_col + w_col;
						val += data_col[data_col_index];
					}
				}
			}
			data_im[index] = val;
		}
	}

	void col2im_gpu(const StorageT* data_col, const int channels,
		const int height, const int width, const int kernel_h, const int kernel_w,
		const int pad_h, const int pad_w, const int stride_h,
		const int stride_w, const int dilation_h, const int dilation_w,
		StorageT* data_im) {
		int height_col = (height + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) /
			stride_h + 1;
		int width_col = (width + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) /
			stride_w + 1;
		int num_kernels = channels * height * width;
		// To avoid involving atomic operations, we will launch one kernel per
		// bottom dimension, and then in the kernel add up the top dimensions.
		// NOLINT_NEXT_LINE(whitespace/operators)
		col2im_gpu_kernel << <FECNN_GET_BLOCKS(num_kernels),
			FECNN_CUDA_NUM_THREADS >> > (
				num_kernels, data_col, height, width, channels, kernel_h, kernel_w,
				pad_h, pad_w, stride_h, stride_w, dilation_h, dilation_w,
				height_col, width_col, data_im);
	}

}

IConvolutionLayer::IConvolutionLayer()
{
	m_pstBias = NULL;
	m_pstWeights = NULL;
	m_bHasBias = false;
	//�������
	m_iGroup = 1;
	m_stDilation = DimsHW(1, 1);	//�˿�ȷ������������
	m_stStride = DimsHW(1, 1);	//��ȷ��򲽳�
	m_stPadding = DimsHW(0, 0);	//��ȷ������������

								//m_iChannelAxis	= 0;			//ͨ��������ά�ȣ�������չ�ã�
}
IConvolutionLayer::IConvolutionLayer(int _nbOutputMaps, Dims _kernelSize, Weights _kernelWeights, Weights _biasWeights)
{
	m_pstBias = NULL;
	m_pstWeights = NULL;
	m_bHasBias = false;

	if (_biasWeights.count > 0 && _biasWeights.values != NULL)
	{
		m_bHasBias = true;
		m_pstBias = (Weights*)malloc(sizeof(Weights));		//Bias����

		if (NULL == m_pstBias)
		{
			printf("CNN_ConvLayer m_pstBias malloc error\n!");
			return;
		}
		m_pstBias->type = _biasWeights.type;
		m_pstBias->count = _biasWeights.count;

		int iTypeSize = Get_Type_Szie(_biasWeights.type);

		if (0 == iTypeSize)
		{
			return;
		}

		m_pstBias->values = CNN_GPU_MemMaloc(0, _biasWeights.count * iTypeSize);

		if (NULL == m_pstBias->values)
		{
			printf("CNN_ConvLayer m_pstBias->values malloc error\n!");
			return;
		}

		int iStatus = CNN_GPU_Memcpy(_biasWeights.count * iTypeSize, _biasWeights.values, (void*)(m_pstBias->values));

		if (iStatus != 0)
		{
			printf("CNN_ConvLayer m_pstBias->values CNN_GPU_Memcpy error\n!");
			return;
		}
	}
	if (_kernelWeights.count < 1 && _kernelWeights.values == NULL)
	{
		printf("CNN_ConvLayer kernelWeights error\n!");
		return;
	}
	m_pstWeights = (Weights*)malloc(sizeof(Weights));		//����

	if (NULL == m_pstWeights)
	{
		printf("CNN_ConvLayer m_pstWeights malloc error\n!");
		return;
	}
	m_pstWeights->type = _kernelWeights.type;
	m_pstWeights->count = _kernelWeights.count;

	int iTypeSize = Get_Type_Szie(_biasWeights.type);

	if (0 == iTypeSize)
	{
		return;
	}

	m_pstWeights->values = CNN_GPU_MemMaloc(0, _kernelWeights.count * iTypeSize);

	if (NULL == m_pstWeights->values)
	{
		printf("CNN_ConvLayer m_pstWeights->values malloc error\n!");
		return;
	}

	int iStatus = CNN_GPU_Memcpy(_kernelWeights.count * iTypeSize, _kernelWeights.values, (void*)(m_pstWeights->values));

	if (iStatus != 0)
	{
		printf("CNN_ConvLayer m_pstWeights->values CNN_GPU_Memcpy error\n!");
		return;
	}

	m_nbOutputMaps = _nbOutputMaps;
	m_stKernel = _kernelSize;

	//�������
	m_iGroup = 0;
	m_stDilation = DimsHW(1, 1);	//�˿�ȷ������������
	m_stStride = DimsHW(1, 1);	//��ȷ��򲽳�
	m_stPadding = DimsHW(0, 0);	//��ȷ������������

	//m_iChannelAxis	= 0;			//ͨ��������ά�ȣ�������չ�ã�
}

IConvolutionLayer::~IConvolutionLayer()
{
	if (NULL != m_pstBias)
	{
		if (NULL != m_pstBias->values)
		{
			CNN_GPU_MemFree((void*)(m_pstBias->values));
			m_pstBias->values = NULL;
		}
		free(m_pstBias);
		m_pstBias = NULL;
	}
	if (NULL != m_pstWeights)
	{
		if (NULL != m_pstWeights->values)
		{
			CNN_GPU_MemFree((void*)(m_pstWeights->values));
			m_pstWeights->values = NULL;
		}
		free(m_pstWeights);
		m_pstWeights = NULL;
	}
}

int IConvolutionLayer::forward(void* _pInData, Dims _stInPut, void* _pOutData, Dims &_stOutPut)
{
	hipdnnHandle_t handle;
	hipdnnCreate(&handle);

	_stOutPut.nbDims = _stInPut.nbDims;
	_stOutPut.d[0] = _stInPut.d[0];//n
	_stOutPut.d[1] = m_stKernel.d[0];	// c 
	int iFeatMap_h = _stOutPut.d[2] = 1 + (_stInPut.d[2] + 2 * m_stPadding.d[0] - m_stKernel.d[2]) / m_stStride.d[0]; //h
	int iFeatMap_w = _stOutPut.d[3] = 1 + (_stInPut.d[3] + 2 * m_stPadding.d[1] - m_stKernel.d[3]) / m_stStride.d[1]; //w

	hipdnnTensorDescriptor_t input_descriptor;
	hipdnnCreateTensorDescriptor(&input_descriptor);
	hipdnnSetTensor4dDescriptor(input_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, _stInPut.d[0], _stInPut.d[1], _stInPut.d[2], _stInPut.d[3]);

	hipdnnTensorDescriptor_t output_descriptor;
	hipdnnCreateTensorDescriptor(&output_descriptor);
	hipdnnSetTensor4dDescriptor(output_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, _stOutPut.d[0], _stOutPut.d[1], _stOutPut.d[2], _stOutPut.d[3]);

	hipdnnFilterDescriptor_t kernel_descriptor;
	hipdnnCreateFilterDescriptor(&kernel_descriptor);
	hipdnnSetFilter4dDescriptor(kernel_descriptor, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, m_stKernel.d[0], m_stKernel.d[1], m_stKernel.d[2], m_stKernel.d[3]);
	// convolution descriptor

	hipdnnConvolutionDescriptor_t conv_descriptor;
	hipdnnCreateConvolutionDescriptor(&conv_descriptor);
	hipdnnSetConvolution2dDescriptor(conv_descriptor,
		m_stPadding.d[0], m_stPadding.d[1], // zero-padding
		m_stStride.d[0], m_stStride.d[1], // stride
		1, 1,
		HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT);

	hipdnnStatus_t eStatus = HIPDNN_STATUS_SUCCESS;
	hipdnnConvolutionFwdAlgo_t algo;
	// algorithm
#if CUDNN_VERSION_MIN(8, 0, 0)
	int returnedAlgoCount = 0;
	int requestedAlgoCount = 1;
	hipdnnConvolutionFwdAlgoPerf_t fwd_algoPer;

	eStatus = cudnnGetConvolutionForwardAlgorithm_v7(handle, input_descriptor,
		kernel_descriptor, conv_descriptor, output_descriptor, requestedAlgoCount,
		&returnedAlgoCount, &fwd_algoPer);
	algo = fwd_algoPer.algo;

	algo = fwd_algoPer.algo;

	if (HIPDNN_STATUS_SUCCESS != eStatus)
	{
		printf("cudnnGetConvolutionForwardAlgorithm_v7 error code:%d\n", eStatus);
		return -1;
	}

#else
	// choose forward and backward algorithms + workspace(s)
	eStatus = hipdnnGetConvolutionForwardAlgorithm(*((hipdnnHandle_t*)pstGlobalInfos->pCuDNNHandles), pInputDesc,
		pstCudnnConv->pFilterDesc, pConvDesc, pOutputDesc, HIPDNN_CONVOLUTION_FWD_SPECIFY_WORKSPACE_LIMIT,
		workspace_limit_bytes, &pstCudnnConv->fwd_algo_[i]);
#endif

	// workspace size && allocate memory
	size_t workspace_size = 0;
	eStatus = hipdnnGetConvolutionForwardWorkspaceSize(handle, input_descriptor, kernel_descriptor, conv_descriptor, output_descriptor, algo, &workspace_size);

	if (HIPDNN_STATUS_SUCCESS != eStatus)
	{
		printf("hipdnnGetConvolutionForwardWorkspaceSize error code:%d\n!", eStatus);
		return -2;
	}

	void * workspace = nullptr;
	hipMalloc(&workspace, workspace_size);

	// convolution
	auto alpha = 1.0f, beta = 0.0f;
	eStatus = hipdnnConvolutionForward(handle, &alpha, input_descriptor, _pInData, kernel_descriptor, m_pstWeights->values,
		conv_descriptor, algo, workspace, workspace_size, &beta, output_descriptor, _pOutData);

	if (HIPDNN_STATUS_SUCCESS != eStatus)
	{
		printf("hipdnnConvolutionForward error code:%d\n!", eStatus);
		return -3;
	}

	if (true == m_bHasBias)
	{
		hipdnnTensorDescriptor_t			bias_descriptor;
		hipdnnCreateTensorDescriptor(&bias_descriptor);
		//hipdnnSetTensor4dDescriptorEx(bias_descriptor, HIPDNN_DATA_FLOAT, _stInPut.d[0], _stInPut.d[1], 1, 1, _stInPut.d[0], _stInPut.d[1], 1, 1);
		eStatus = setTensor4dDesc<float>(&bias_descriptor, 1, m_pstBias->count, 1, 1);
		auto alpha = 1.0f, beta = 1.0f;
		eStatus = hipdnnAddTensor(handle, &alpha, bias_descriptor, m_pstBias->values, &beta, output_descriptor, _pOutData);

		if (HIPDNN_STATUS_SUCCESS != eStatus)
		{
			printf("hipdnnAddTensor error code:%d\n!", eStatus);
			return -4;
		}

		hipdnnDestroyTensorDescriptor(bias_descriptor);
	}

	hipFree(workspace);

	hipdnnDestroyTensorDescriptor(input_descriptor);
	hipdnnDestroyTensorDescriptor(output_descriptor);
	hipdnnDestroyConvolutionDescriptor(conv_descriptor);
	hipdnnDestroyFilterDescriptor(kernel_descriptor);
	hipdnnDestroy(handle);

	return 0;
}




hipblasStatus_t CNN_Util_Math_Gemm_GPU(const CBLAS_TRANSPOSE TransA,
	const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
	const float alpha, const float* A, const float* B, const float beta, float* C, hipblasHandle_t _hCuBLAS)
{
	// Note that cublas follows fortran order.
	int lda = (TransA == CblasNoTrans) ? K : M;
	int ldb = (TransB == CblasNoTrans) ? N : K;
	hipblasOperation_t cuTransA = (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
	hipblasOperation_t cuTransB = (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
	return hipblasSgemm(_hCuBLAS, cuTransB, cuTransA, N, M, K, &alpha, B, ldb, A, lda, &beta, C, N);
}


template <typename Dtype>
__global__ void set_kernel(const int n, const Dtype alpha, Dtype* y)
{
	CUDA_KERNEL_LOOP(index, n)
	{
		y[index] = alpha;
	}
}

template <typename Dtype>
void CNN_Util_Math_Set_GPU(const int N, const Dtype alpha, Dtype* Y)
{
	if (alpha == 0)
	{
		hipError_t eErr = hipMemset(Y, 0, sizeof(Dtype) * N);  // NOLINT(caffe/alt_fn)
		return;
	}
	// NOLINT_NEXT_LINE(whitespace/operators)
	set_kernel<Dtype> << <CNN_GET_BLOCKS(N), CNN_CUDA_NUM_THREADS >> >(N, alpha, Y);

	hipDeviceSynchronize();
}

template void CNN_Util_Math_Set_GPU<int>(const int N, const int alpha, int* Y);
template void CNN_Util_Math_Set_GPU<float>(const int N, const float alpha, float* Y);
template void CNN_Util_Math_Set_GPU<double>(const int N, const double alpha, double* Y);

int IConvolutionLayer::forwardGMM(void* _pInData, Dims _stInPut, void* _pOutData, 
	Dims &_stOutPut, void *_pBiasMultip, void *_pBuffer)
{
#if 1
	_stOutPut.nbDims = _stInPut.nbDims;
	_stOutPut.d[0] = _stInPut.d[0];//n
	_stOutPut.d[1] = m_stKernel.d[0];	// c 
	int iFeatMap_h = _stOutPut.d[2] = 1 + (_stInPut.d[2] + 2 * m_stPadding.d[0] - m_stKernel.d[2]) / m_stStride.d[0]; //h
	int iFeatMap_w = _stOutPut.d[3] = 1 + (_stInPut.d[3] + 2 * m_stPadding.d[1] - m_stKernel.d[3]) / m_stStride.d[1]; //w


	int M = m_stKernel.d[0];
	int N = iFeatMap_h * iFeatMap_w;
	int K = _stInPut.d[1] * m_stKernel.d[2] * m_stKernel.d[3];
	int iInputStep = _stInPut.d[1] * _stInPut.d[2] * _stInPut.d[3];
	int iOutputStep = M * N;
	bool b1x1 = (1 == m_stKernel.d[2]) && (1 == m_stKernel.d[3]);
	hipblasHandle_t hCuBLAS = NULL;

	hipblasCreate(&hCuBLAS);

	const float* weight = (float*)(m_pstWeights->values);

	int iBuffSize = sizeof(float)* N * m_stKernel.d[2] * m_stKernel.d[3] * _stInPut.d[1] * _stInPut.d[0];

	float *gmm_buf = nullptr;

	for (int n = 0; n < _stInPut.d[0]; ++n)
	{
		float* col_buff = ((float*)_pInData) + n * iInputStep;
		if (!b1x1)
		{
			if (nullptr == gmm_buf) {
				hipMalloc((void**)&gmm_buf, iBuffSize);//
			}
			if (nullptr == gmm_buf)
			{
				printf("forwardGMM hipMalloc gmm_buf error\n");
				continue;
			}

			CNN_Im2Col_GPU<float>(col_buff, _stInPut.d[1], _stInPut.d[2], _stInPut.d[3],
				m_stKernel.d[2], m_stKernel.d[3],
				m_stPadding.d[0], m_stPadding.d[1],
				m_stStride.d[0], m_stStride.d[1],
				m_stDilation.d[0], m_stDilation.d[1],
				(float*)gmm_buf);
			col_buff = (float*)gmm_buf;
		}
		hipblasStatus_t eStatus = CNN_Util_Math_Gemm_GPU(CblasNoTrans, CblasNoTrans, M, N, K,
			1., weight, col_buff, 0., (float*)_pOutData + n * iOutputStep, hCuBLAS);

		if (HIPBLAS_STATUS_SUCCESS != eStatus)
		{
			printf("CNN_Util_Math_Gemm_GPU error:%d\n", eStatus);
		}

		if (m_bHasBias)
		{
			//int iOutSize = pstBasicInfos->iOutputW * pstBasicInfos->iOutputH;
			void * tmp_buf = nullptr;

			hipMalloc((void**)&tmp_buf, N * sizeof(float));//

			CNN_Util_Math_Set_GPU(N, 1.0f, (float*)tmp_buf);

			eStatus = CNN_Util_Math_Gemm_GPU(CblasNoTrans, CblasNoTrans, M, N, 1,
				1., (float*)(m_pstBias->values), (float*)tmp_buf, 1., (float*)_pOutData + n * iOutputStep, hCuBLAS);

			if (HIPBLAS_STATUS_SUCCESS != eStatus)
			{
				printf("CNN_Util_Math_Gemm_GPU error:%d\n", eStatus);
			}

			hipFree(tmp_buf);
		}
	}
	if (gmm_buf != nullptr)
	{
		hipFree(gmm_buf);
	}
	hipblasDestroy(hCuBLAS);
	return 0;
#else
	_stOutPut.nbDims = _stInPut.nbDims;
	_stOutPut.d[0] = _stInPut.d[0];//n
	_stOutPut.d[1] = m_stKernel.d[0];	// c 
	int iFeatMap_h = _stOutPut.d[2] = 1 + (_stInPut.d[2] + 2 * m_stPadding.d[0] - m_stKernel.d[2]) / m_stStride.d[0]; //h
	int iFeatMap_w = _stOutPut.d[3] = 1 + (_stInPut.d[3] + 2 * m_stPadding.d[1] - m_stKernel.d[3]) / m_stStride.d[1]; //w

	int M = m_stKernel.d[0];
	int N = iFeatMap_h * iFeatMap_w;
	int K = _stInPut.d[1] * m_stKernel.d[2] * m_stKernel.d[3];
	fecnn::StorageT* weights = (fecnn::StorageT*)(m_pstWeights->values);

	fecnn::StorageT* col_buff = nullptr;
	int iBuffSize = sizeof(fecnn::StorageT)* N * m_stKernel.d[2] * m_stKernel.d[3] * _stInPut.d[1] * _stInPut.d[0];
	//float *gmm_buf = nullptr;
	
	if (nullptr == col_buff) {
		hipMalloc((void**)&col_buff, iBuffSize);//
	}
	if (nullptr == col_buff)
	{
		printf("forwardGMM hipMalloc gmm_buf error\n");
		return -2;
	}
	fecnn::im2col_gpu((fecnn::StorageT*)_pInData, _stInPut.d[1], _stInPut.d[2], _stInPut.d[3],
		m_stKernel.d[2], m_stKernel.d[3],
		m_stPadding.d[0], m_stPadding.d[1],
		m_stStride.d[0], m_stStride.d[1],
		1, 1, col_buff);

	hipblasHandle_t hCuBLAS = NULL;

	hipblasCreate(&hCuBLAS);

	fecnn::fecnn_gpu_gemm(hCuBLAS, HIPBLAS_OP_N, HIPBLAS_OP_T,
		M, N, K,
		(fecnn::StorageT)1., (fecnn::StorageT*)_pOutData,
		col_buff,
		(fecnn::StorageT)1., weights);

	void * bias_multGPU = nullptr;

	hipMalloc((void**)&bias_multGPU, N * sizeof(fecnn::StorageT));//

	CNN_Util_Math_Set_GPU(N, 1.0f, (fecnn::StorageT*)bias_multGPU);

	fecnn::fecnn_gpu_gemm(hCuBLAS, HIPBLAS_OP_N, HIPBLAS_OP_N,
		M, N, 1,
		(fecnn::StorageT)1., (fecnn::StorageT*)(m_pstBias->values), (fecnn::StorageT*)bias_multGPU,
		(fecnn::StorageT)1., (fecnn::StorageT*)_pOutData);

	hipFree(bias_multGPU);

	if (col_buff != nullptr)
	{
		hipFree(col_buff);
	}
	hipblasDestroy(hCuBLAS);
	return 0;
#endif // 0
}

